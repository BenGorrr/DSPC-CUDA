#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <ctime>

#define N 2048
#define IT 3

__global__ void JacobiIteration(int n, float* a, float* b, float* x, float* x_result) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        float sigma = 0;
        for (int j = 0; j < n; j++) {
            if (j != i) {
                sigma += a[i + j * n] * x[j];
            }
        }
        x_result[i] = (b[i] - sigma) / a[i + i * n];
    }
}

__global__ void initAMatrix(int n, float* a) {
    int i;
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < n; j += blockDim.x * gridDim.x) {

        for (i = 0; i < n; i++) {
            if (j == i - 1) {
                a[j + i * n] = -1.0;
            }
            else if (j == i) {
                a[j + i * n] = 2.0;
            }
            else if (j == i + 1) {
                a[j + i * n] = -1.0;
            }
            else {
                a[j + i * n] = 0.0;
            }
        }
    }

}

__global__ void copy(int n, float* a, float* b) {
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < n; j += blockDim.x * gridDim.x) {
        a[j] = b[j];
    }

}

void swap(float*& a, float*& b) {
    float* temp = a;
    a = b;
    b = temp;
}



int main() {
    float* a, * b, * x, * x_result;

    // alloc
    hipMallocManaged(&a, N * N * sizeof(float));
    hipMallocManaged(&b, N * sizeof(float));
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&x_result, N * sizeof(float));

    // init
    for (int i = 0; i < N; i++)
    {
        b[i] = 3.0;
    }
    b[N - 1] = (float)(N + 1);

    for (int i = 0; i < N; i++)
    {
        x[i] = 0.0;
    }

    initAMatrix << <32, 32 >> > (N, a);

    std::clock_t start;
    double duration;

    start = std::clock();

    for (int it = 0; it < IT; it++) {

        JacobiIteration << <32, 32 >> > (N, a, b, x, x_result);
        hipDeviceSynchronize();
        swap(x, x_result);

    }

    duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;

    //std::cout << "Process time: " << duration << "s" << std::endl;

    for (int i = 0; i < N; i++) {
        printf("%f ", x[i]);
    }

    printf("Process time: %.4f s\n", duration);

    return 0;
}